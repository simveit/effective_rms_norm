#include <hip/hip_runtime.h>

#include <iostream>
#include <random>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, char const *func, char const *file, int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(char const *file, int line) {
  hipError_t const err{hipGetLastError()};
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

// x is [token, hiddenDim], each block handles one token.
template <int hiddenDim, int threadsPerBlock>
__global__ void rmsNormKernelSmem(float *x, float *w, float eps, float *y) {
  __shared__ float squaredPerThread[threadsPerBlock];
  __shared__ float xShared[hiddenDim];
  __shared__ float rms_;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;

  float sum = 0.0f;

  for (int i = tid; i < hiddenDim; i += threadsPerBlock) {
    int index = bid * hiddenDim + i;
    float x_ = x[index];
    xShared[i] = x_;
    sum += x_ * x_;
  }
  squaredPerThread[tid] = sum;
  __syncthreads();

  for (int activeThreads = threadsPerBlock / 2; activeThreads > 0;
       activeThreads >>= 1) {
    if (tid < activeThreads) {
      squaredPerThread[tid] += squaredPerThread[tid + activeThreads];
    }
    __syncthreads();
  }

  if (tid == 0) {
    rms_ = rsqrtf(squaredPerThread[tid] / hiddenDim + eps);
  }
  __syncthreads();

  for (int i = tid; i < hiddenDim; i += threadsPerBlock) {
    float val = xShared[i] * rms_ * w[i];
    y[bid * hiddenDim + i] = val;
  }
}

template <int numTokens, int hiddenDim, int threadsPerBlock>
void launchRmsNormSmem(float *x, float *w, float eps, float *y) {
  rmsNormKernelSmem<hiddenDim, threadsPerBlock>
      <<<numTokens, threadsPerBlock>>>(x, w, eps, y);
}

template <int numTokens, int hiddenDim>
void launchRmsNormCpu(float *x, float *w, float eps, float *y) {
  float rms;
  for (int token = 0; token < numTokens; token++) {
    rms = 0;
    for (int hidden = 0; hidden < hiddenDim; hidden++) {
      rms += x[token * hiddenDim + hidden] * x[token * hiddenDim + hidden];
    }
    rms = sqrt(rms / hiddenDim + eps);
    for (int hidden = 0; hidden < hiddenDim; hidden++) {
      y[token * hiddenDim + hidden] =
          x[token * hiddenDim + hidden] / rms * w[hidden];
    }
  }
}

int main() {
  const int numTokens = 1 << 18;
  const int hiddenDim = 1 << 12;
  const size_t size = numTokens * hiddenDim * sizeof(float);
  const int threadsPerBlock = 1 << 9;

  float *xHost = new float[numTokens * hiddenDim];
  float *wHost = new float[hiddenDim];
  float eps = 1e-5f;
  float *yHost = new float[numTokens * hiddenDim];
  float *yReference = new float[numTokens * hiddenDim];

  std::default_random_engine generator(42);
  std::normal_distribution<float> distribution(0.0, 1.0);

  for (int i = 0; i < numTokens * hiddenDim; i++) {
    xHost[i] = distribution(generator);
    if (i < hiddenDim) {
      wHost[i] = 1.0f;
    }
  }

  float *xDevice;
  float *wDevice;
  float *yDevice;

  CHECK_CUDA_ERROR(hipMalloc(&xDevice, size));
  CHECK_CUDA_ERROR(hipMalloc(&wDevice, size / numTokens));
  CHECK_CUDA_ERROR(hipMalloc(&yDevice, size));

  CHECK_CUDA_ERROR(hipMemcpy(xDevice, xHost, size, hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(
      hipMemcpy(wDevice, wHost, size / numTokens, hipMemcpyHostToDevice));

  launchRmsNormSmem<numTokens, hiddenDim, threadsPerBlock>(xDevice, wDevice,
                                                           eps, yDevice);

  CHECK_CUDA_ERROR(hipMemcpy(yHost, yDevice, size, hipMemcpyDeviceToHost));
  CHECK_LAST_CUDA_ERROR();

  launchRmsNormCpu<numTokens, hiddenDim>(xHost, wHost, eps, yReference);

  for (int token = 0; token < numTokens; token++) {
    for (int hidden = 0; hidden < hiddenDim; hidden++) {
      float y = yHost[token * hiddenDim + hidden];
      float yR = yReference[token * hiddenDim + hidden];

      if (fabs(y - yR) > 1e-3) {
        std::cout << "Error at token = " << token << " , hidden = " << hidden
                  << std::endl;
        std::cout << "y = " << y << " , yR = " << yR << std::endl;
        return 1;
      }
    }
  }
  std::cout << "Verification successfull" << std::endl;

  int numRounds = 10000;
  size_t numCrossMemoryBound = 2 * size;
  hipEvent_t start, stop;
  float time;

  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  CHECK_CUDA_ERROR(hipEventRecord(start));
  for (int i = 0; i < numRounds; i++) {
    launchRmsNormSmem<numTokens, hiddenDim, threadsPerBlock>(xDevice, wDevice,
                                                             eps, yDevice);
  }
  CHECK_CUDA_ERROR(hipEventRecord(stop));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
  CHECK_LAST_CUDA_ERROR();

  float latency = time / numRounds;
  float bandwidth = (numCrossMemoryBound / latency) / 1e6;

  std::cout << "Latency = " << latency << " ms" << std::endl;
  std::cout << "Bandwidth = " << bandwidth << " GB/s" << std::endl;
  std::cout << "% of max = " << bandwidth / 3300 * 100 << " %" << std::endl;

  CHECK_CUDA_ERROR(hipFree(xDevice));
  CHECK_CUDA_ERROR(hipFree(wDevice));
  CHECK_CUDA_ERROR(hipFree(yDevice));

  free(xHost);
  free(wHost);
  free(yHost);

  return 0;
}